#include "hip/hip_runtime.h"
#include "raycast.cuh"

stack_t* new_stack(){
    stack_t* stack = (stack_t*)malloc(sizeof(stack_t));
    stack->size = 0;
    stack->buffer_size = 1024;
    stack->pixels = (int3*)malloc(sizeof(int3)*1024);

    return stack;
}

void push(stack_t* stack, int3 p){
    if(stack->size == stack->buffer_size){
        stack->buffer_size *= 2;
        int3* temp = stack->pixels;
        stack->pixels = (int3*)malloc(sizeof(int3)*stack->buffer_size);
        memcpy(stack->pixels, temp, sizeof(int3)*stack->buffer_size/2);
        free(temp);

    }
    stack->pixels[stack->size] = p;
    stack->size += 1;
}

int3 pop(stack_t* stack){
    stack->size -= 1;
    return stack->pixels[stack->size];
}

// float3 utilities
float3 cross(float3 a, float3 b){
    float3 c;
    c.x = a.y*b.z - a.z*b.y;
    c.y = a.z*b.x - a.x*b.z;
    c.z = a.x*b.y - a.y*b.x;

    return c;
}

float3 normalize(float3 v){
    float l = sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
    v.x /= l;
    v.y /= l;
    v.z /= l;

    return v;
}

float3 add(float3 a, float3 b){
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;

    return a;
}

float3 scale(float3 a, float b){
    a.x *= b;
    a.y *= b;
    a.z *= b;

    return a;
}

void setCudaDevice(hipDeviceProp_t* p, int device){
    gEC(hipSetDevice(device));
    gEC(hipGetDeviceProperties(p, device));
    gEC(hipDeviceSynchronize());
}

int getAmountOfSMs(int device){
    hipDeviceProp_t p;
    setCudaDevice(&p, device);
    return p.multiProcessorCount;
}

int getThreadsPerBlock(int device){
    hipDeviceProp_t p;
    setCudaDevice(&p, device);
    return p.maxThreadsDim[0];
}

int getBlocksPerSM(int device, int dim){
    if(2 < dim){
        fprintf(stderr, "Not enough block dimensions in SM!! dim: %d\n", dim);
        exit(-1);
    }
    hipDeviceProp_t p;
    setCudaDevice(&p, device);
    return p.maxGridSize[dim];
}

int getMaxThreadsPerSM(int device){
    hipDeviceProp_t p;
    setCudaDevice(&p, device);
    return p.maxThreadsPerMultiProcessor;
}

// Prints CUDA device properties
void print_properties(){
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    printf("Device count: %d\n\n", deviceCount);

    hipDeviceProp_t p;
    for (int i = 0; i < deviceCount; ++i){
        int cudaReturnStatus = hipSetDevice(i);
        if (hipSuccess != cudaReturnStatus){
            printf("hipSetDevice(%d) returned error\n", i);
            continue;
        }

        cudaReturnStatus = hipGetDeviceProperties (&p, i);
        if (hipSuccess != cudaReturnStatus){
            printf("hipGetDeviceProperties(&p, %d) returned error\n", i);
            continue;
        }

        //If all went well, print info:
        printf("Device #%d, Name: %s\n" , (i+1), p.name);
        printf("Compute capability: %d.%d\n", p.major, p.minor);

        printf("#Threads per Warp: %d\n", p.warpSize);

        printf("Multiprocessor (SM/SMX) count: %d\n", p.multiProcessorCount);

        printf("Max #threads per SM/SMX: %d\n", p.maxThreadsPerMultiProcessor);

        printf("Total memory: %zd MiB \nShared memory per thread block in a SM/SMX: %zd KiB\n",
            p.totalGlobalMem/(1024*1024), p.sharedMemPerBlock/1024);

        printf("Max #registers per Block: %d\n", p.regsPerBlock);

        printf("Max threads per Blocks: ");
        for (int j = 0; j < 2; ++j){
            printf("%d, ", p.maxThreadsDim[j]);
        }printf("%d\n", p.maxThreadsDim[2]);

        printf("Max Grid Size: ");
        for (int j = 0; j < 2; ++j){
            printf("%d, ", p.maxGridSize[j]);
        }printf("%d\n", p.maxGridSize[2]);

        printf("Max Threads per Block: %d\n", p.maxThreadsPerBlock);

        printf("Are concurrent kernels supported?: %s\n",
            p.concurrentKernels ? "yes" : "no");

        if (p.asyncEngineCount){
            printf("Device can transfer data %s host and device while executing a kernel\n",
                p.asyncEngineCount == 2 ? "both ways between" : "one way between");
        } else{
            printf("Device cannot transfer data between host/device while a kernel is running\n");
        }

        printf("\n");
    }
}

// Fills data with values
unsigned char func(int x, int y, int z){
    unsigned char value = rand() % 20;

    int x1 = 300;
    int y1 = 400;
    int z1 = 100;
    float dist = sqrt((x-x1)*(x-x1) + (y-y1)*(y-y1) + (z-z1)*(z-z1));

    if(dist < 100){
        value  = 30;
    }

    x1 = 100;
    y1 = 200;
    z1 = 400;
    dist = sqrt((x-x1)*(x-x1) + (y-y1)*(y-y1) + (z-z1)*(z-z1));

    if(dist < 50){
        value = 50;
    }

    if(x > 200 && x < 300 && y > 300 && y < 500 && z > 200 && z < 300){
        value = 45;
    }
    if(x > 0 && x < 100 && y > 250 && y < 400 && z > 250 && z < 400){
        value =35;
    }
    return value;
}

unsigned char* create_data(){
    unsigned char* data = (unsigned char*)malloc(sizeof(unsigned char) * DATA_DIM*DATA_DIM*DATA_DIM);

    for(int i = 0; i < DATA_DIM; i++){
        for(int j = 0; j < DATA_DIM; j++){
            for(int k = 0; k < DATA_DIM; k++){
                data[i*DATA_DIM*DATA_DIM + j*DATA_DIM + k]= func(k,j,i);
            }
        }
    }

    return data;
}

// Checks if position is inside the volume (float3 and int3 versions)
int inside(float3 pos){
    int x = (pos.x >= 0 && pos.x < DATA_DIM-1);
    int y = (pos.y >= 0 && pos.y < DATA_DIM-1);
    int z = (pos.z >= 0 && pos.z < DATA_DIM-1);

    return x && y && z;
}

int inside(int3 pos){
    int x = (pos.x >= 0 && pos.x < DATA_DIM);
    int y = (pos.y >= 0 && pos.y < DATA_DIM);
    int z = (pos.z >= 0 && pos.z < DATA_DIM);

    return x && y && z;
}

int index(int3 pos){
    return pos.z*DATA_DIM*DATA_DIM
            + pos.y*DATA_DIM + pos.x;
}

int getKernelThreadAmount(dim3** sizes){
    return (sizes[0]->x*sizes[0]->y*sizes[0]->z) *
            (sizes[1]->x*sizes[1]->y*sizes[1]->z);
}

// Indexing function (note the argument order)
int index(int z, int y, int x){
    return z * DATA_DIM*DATA_DIM + y*DATA_DIM + x;
}

// Trilinear interpolation
float value_at(float3 pos, unsigned char* data){
    if(!inside(pos)){
        return 0;
    }

    int x = floor(pos.x);
    int y = floor(pos.y);
    int z = floor(pos.z);

    int x_u = ceil(pos.x);
    int y_u = ceil(pos.y);
    int z_u = ceil(pos.z);

    float rx = pos.x - x;
    float ry = pos.y - y;
    float rz = pos.z - z;

    float a0 = rx*data[index(z,y,x)] + (1-rx)*data[index(z,y,x_u)];
    float a1 = rx*data[index(z,y_u,x)] + (1-rx)*data[index(z,y_u,x_u)];
    float a2 = rx*data[index(z_u,y,x)] + (1-rx)*data[index(z_u,y,x_u)];
    float a3 = rx*data[index(z_u,y_u,x)] + (1-rx)*data[index(z_u,y_u,x_u)];

    float b0 = ry*a0 + (1-ry)*a1;
    float b1 = ry*a2 + (1-ry)*a3;

    float c0 = rz*b0 + (1-rz)*b1;

    return c0;
}

// Check if two values are similar, threshold can be changed.
int similar(unsigned char* data, int3 a, int3 b){
    unsigned char va = data[index(a)];
    unsigned char vb = data[index(b)];

    int i = abs(va-vb) < 1;
    return i;
}

void gpuAssert(hipError_t code, const char *file, int line, int abort){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s, @%s:%d\n", hipGetErrorString(code), file, line);
      if(abort){
        exit(code);
      }
   }
}

void createCudaEvent(hipEvent_t* event){
    gEC(hipEventCreate(event)); gEC(hipEventRecord(*event, 0));
    gEC(hipEventSynchronize(*event)); return;
}

float getCudaEventTime(hipEvent_t start, hipEvent_t end){
    float result = -1;
    gEC(hipEventElapsedTime(&result, start, end));
    return result;
}

dim3** getGridAndBlockSize(int device){
    dim3 grid, block;
    int SMs = getAmountOfSMs(device),
        totThreads = getMaxThreadsPerSM(device)*SMs,
        totBlocks = totThreads/512; //Ref amount threads per block below
    dim3 **sizes = (dim3**) malloc(sizeof(dim3*)*2);
    sizes[0] = (dim3*) malloc(sizeof(dim3));
    sizes[1] = (dim3*) malloc(sizeof(dim3));
    printf("Done assigning to size!\n");

    //Hardcoding blockdim values (8^3 = 512 = DATA_DIM)
    block.x = 8; grid.x = 1;
    block.y = 8; grid.y = 1;
    block.z = 8; grid.z = 1;

    while(8 <= totBlocks){
        grid.x += 2;
        grid.y += 2;
        grid.z += 2;
        totBlocks /= 8;
    }
    /*if (totBlocks){
        printf("\t\t%d threads left, making %d blocks\n", totBlocks*512, totBlocks);
    }*/
    printf("Done assigning grid and block values!\n");

    printf("Before memcpy's!\n");
    memcpy(sizes[0], &grid, sizeof(dim3));
    memcpy(sizes[1], &block, sizeof(dim3));
    printf("Done with memcpy!\n");
    return sizes;
}

/*################# Functions accessible by kernels ##############
__device__ int getBlockId_3D(){
    return blockIdx.x + (blockIdx.y*gridDim.x)
            + (blockIdx.z*gridDim.x*gridDim.y);
}

__device__ int gpu_getDataIndex(int3 pos){
    return pos.z*IMAGE_SIZE
        + pos.y*DATA_DIM + pos.x;
}

__device__ int gpu_isPosInside(int3 pos){
    int x = (pos.x >= 0 && pos.x < DATA_DIM-1);
    int y = (pos.y >= 0 && pos.y < DATA_DIM-1);
    int z = (pos.z >= 0 && pos.z < DATA_DIM-1);
    return x && y && z;
}

__device__ int getBlockThreadId_3D(){
    return threadIdx.x + (threadIdx.y*blockDim.x)
            + (threadIdx.z*blockDim.x*blockDim.y);
}

__device__ int getGlobalIdx_3D_3D(){
    int blockId = getBlockId_3D();
    int threadId = getBlockThreadId_3D() +
            blockId*(blockDim.x*blockDim.y*blockDim.z);
    return threadId;
}

__device__ int3 getGlobalPos(int globalThreadId){
    int3 pos = {
        .x = globalThreadId,
            .y = 0, .z = 0};

    //Check if x > (512^2 - 1)
    if ((IMAGE_SIZE-1) < pos.x){
        pos.z = pos.x/IMAGE_SIZE;
        pos.x -= pos.z*IMAGE_SIZE;
    }

    //Check if x > (512 - 1)
    if ((IMAGE_DIM-1) < pos.x){
        pos.y = pos.x/IMAGE_DIM;
        pos.x -= pos.y*IMAGE_DIM;
    }

    return pos;
}

__device__ int gpu_similar(unsigned char* data, int3 a, int3 b){
    unsigned char va = data[gpu_getDataIndex(a)];
    unsigned char vb = data[gpu_getDataIndex(b)];
    return (abs(va-vb) < 1);
}
//##############*/
