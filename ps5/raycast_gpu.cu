#include "hip/hip_runtime.h"
#include "raycast.h"

__global__ void region_grow_kernel(unsigned char* data, unsigned char* region, int* finished){
    *finished = 1;
    return;
}

__global__ void raycast_kernel(unsigned char* data, unsigned char* image, unsigned char* region){
    //blah
}

unsigned char* grow_region_gpu(unsigned char* data){
    int finished = 0, *gpu_finished;
    hipEvent_t start, end;
    dim3 blockDim, gridDim;
    unsigned char* cudaImage, *cudaRegion, *region;
    blockDim.x = 512, gridDim.x = 512, gridDim.y = 512;
    region = (unsigned char*) calloc(IMAGE_SIZE, sizeof(unsigned char));

    //Malloc image on cuda device
    gEC(hipMalloc(&cudaImage, sizeof(unsigned char)*IMAGE_SIZE));
    //Malloc region on cuda device
    gEC(hipMalloc(&cudaRegion, sizeof(unsigned char)*IMAGE_SIZE));
    gEC(hipMalloc(&gpu_finished, sizeof(int)));

    //Copy image and region over to device
    createCudaEvent(&start);
    gEC(hipMemcpy(cudaImage, data, sizeof(unsigned char)*IMAGE_SIZE, hipMemcpyHostToDevice));
    gEC(hipMemcpy(cudaRegion, region, sizeof(unsigned char)*IMAGE_SIZE, hipMemcpyHostToDevice));
    createCudaEvent(&end);
    printf("Copying image and region to device took %f ms\n",
        getCudaEventTime(start, end));


    while(!finished){
        printf("Entered while-loop\n");
        gEC(hipMemcpy(gpu_finished, &finished, sizeof(int), hipMemcpyHostToDevice));
        region_grow_kernel<<<gridDim, blockDim>>>(data, region, gpu_finished);
        gEC(hipMemcpy(&finished, gpu_finished, sizeof(int), hipMemcpyDeviceToHost));
    }




    //Copy region from device
    createCudaEvent(&start);
    gEC(hipMemcpy(region, cudaRegion, sizeof(unsigned char)*IMAGE_SIZE, hipMemcpyDeviceToHost));
    createCudaEvent(&end);
    printf("Copying region from device took %f ms\n", getCudaEventTime(start, end));

    return region;
}

unsigned char* raycast_gpu(unsigned char* data, unsigned char* region){

    return NULL;
}

int main(int argc, char** argv){
    //float ms_time;
    print_properties();

    printf("Done printing properties\n");

    printf("size of data: %zd\n", sizeof(unsigned char)*DATA_DIM*DATA_DIM*DATA_DIM/(1024*1024));

    unsigned char* data = create_data();

    printf("Done creating data\n");


    unsigned char* region = grow_region_gpu(data);
    //printf("grow_region_gpu() took %f ms\n", ms_time);

    printf("Done creating region\n");

    unsigned char* image = raycast_gpu(data, region);
    /*printf("raycast_gpu() took %f ms\n", ms_time);*/

    printf("Done creating image\n");

    //write_bmp(image, IMAGE_DIM, IMAGE_DIM, "raycast_gpu_out.bmp");

    printf("Done with program\n");
    return 0;
}
