#include "hip/hip_runtime.h"
#include "raycast.cuh"

__device__ int getBlockId_3D(){
    return blockIdx.x + (blockIdx.y*gridDim.x)
            + (blockIdx.z*gridDim.x*gridDim.y);
}

__device__ int gpu_getDataIndex(int3 pos){
    return pos.z*IMAGE_SIZE
        + pos.y*DATA_DIM + pos.x;
}

__device__ int gpu_isPosInside(int3 pos){
    int x = (pos.x >= 0 && pos.x < DATA_DIM-1);
    int y = (pos.y >= 0 && pos.y < DATA_DIM-1);
    int z = (pos.z >= 0 && pos.z < DATA_DIM-1);
    return x && y && z;
}

__device__ int getBlockThreadId_3D(){
    return threadIdx.x + (threadIdx.y*blockDim.x)
            + (threadIdx.z*blockDim.x*blockDim.y);
}

__device__ int getGlobalIdx_3D_3D(){
    int blockId = getBlockId_3D();
    int threadId = getBlockThreadId_3D() +
            blockId*(blockDim.x*blockDim.y*blockDim.z);
    return threadId;
}

__device__ int3 getGlobalPos(int globalThreadId){
    int3 pos = {
        .x = globalThreadId,
            .y = 0, .z = 0};

    //Check if x > (512^2 - 1)
    if ((IMAGE_SIZE-1) < pos.x){
        pos.z = pos.x/IMAGE_SIZE;
        pos.x -= pos.z*IMAGE_SIZE;
    }

    //Check if x > (512 - 1)
    if ((IMAGE_DIM-1) < pos.x){
        pos.y = pos.x/IMAGE_DIM;
        pos.x -= pos.y*IMAGE_DIM;
    }

    return pos;
}

__device__ int gpu_similar(unsigned char* data, int3 a, int3 b){
    unsigned char va = data[gpu_getDataIndex(a)];
    unsigned char vb = data[gpu_getDataIndex(b)];
    return (abs(va-vb) < 1);
}

__global__ void region_grow_kernel(unsigned char* data, unsigned char* region, int* changed){
    *changed = 0;
    const int dx[6] = {-1,1,0,0,0,0};
    const int dy[6] = {0,0,-1,1,0,0};
    const int dz[6] = {0,0,0,0,-1,1};
    int3 pixel = {.x = threadIdx.x,
        .y = blockIdx.x, .z = blockIdx.y};
    int tid = getGlobalIdx_3D_3D();

    /*if(pixel.y == 0 && pixel.x == 0 && pixel.z == 0){
        int x = 2147483648/2;
        printf("Value: 1073741824\n");
        printf("Ints : %d\n", x);
    }*/

    //printf("tid: %d: .x = %d, .y = %d, .z = %d\n", tid, pixel.x, pixel.y, pixel.z);

    if(NEW_VOX == region[tid]){
        printf("Entered first if!\n");
        printf("tid: .x=%d, .y=%d, .z=%d\n", pixel.x, pixel.y, pixel.z);
        int3 pos;
        region[tid] = VISITED;
        for (int i = 0; i < 6; ++i){
            pos = pixel;
            pos.x += dx[i];
            pos.y += dy[i];
            pos.z += dz[i];
            if (//Check that pos pixel is inside image/region
                gpu_isPosInside(pos) &&
                //Check that it's not already been "discovered"
                !region[tid] &&
                //Check that the corresponding color values actually match
                abs(data[tid] - data[gpu_getDataIndex(pos)]) < 1){
                //then
                printf("Found neighbour!\n");
                region[tid] = NEW_VOX;
                atomicAdd(changed, 1);
            }
        }
    }

    //__syncthreads();
    *changed = 1;
    return;
}

void gpuGRKCall(dim3** sizes, int *changed, int* gpu_changed, unsigned char *image, unsigned char* region, int offset){
    int tmp = *changed;
    gEC(hipMemcpy(gpu_changed, changed, sizeof(int), hipMemcpyHostToDevice));
    //printf("Finished changed memcpy to device!\n");
    region_grow_kernel<<<*sizes[0], *sizes[1]>>>(&image[offset], &region[offset], gpu_changed);
    gEC(hipMemcpy(changed, gpu_changed, sizeof(int), hipMemcpyDeviceToHost));
    tmp += *changed;
    *changed = tmp;
}

unsigned char* grow_region_gpu(unsigned char* data){
    printf("\nEntered grow_region_gpu!\n");

    hipEvent_t start, end;
    dim3 **sizes = getGridAndBlockSize(0);
    int3 seed = {.x = 50, .y = 300, .z = 300};
    unsigned char *cudaImage, *cudaRegion, *region;
    region = (unsigned char*) calloc(DATA_SIZE, sizeof(unsigned char));
    region[seed.z*IMAGE_SIZE + seed.y*DATA_DIM + seed.x] = NEW_VOX;
    int changed = 1, *gpu_changed, rounds = DATA_SIZE/getKernelThreadAmount(sizes);
    printf("Done instantiating variables...\n");

    gEC(hipMalloc(&gpu_changed, sizeof(int)));
    //Malloc image on cuda device
    gEC(hipMalloc(&cudaImage, dataSize));
    //Malloc region on cuda device
    gEC(hipMalloc(&cudaRegion, dataSize));

    printf("Done mallocing on CUDA device!\n");

    //Copy image and region over to device
    createCudaEvent(&start);
    gEC(hipMemcpy(cudaImage, data, dataSize, hipMemcpyHostToDevice));
    gEC(hipMemcpy(cudaRegion, region, dataSize, hipMemcpyHostToDevice));
    createCudaEvent(&end);
    printf("Copying image and region to device took %f ms\n",
        getCudaEventTime(start, end));

    printf("grid.x: %d, grid.y: %d, grid.z: %d\n", sizes[0]->x, sizes[0]->y, sizes[0]->z);
    printf("block.x: %d, block.y: %d, block.z: %d\n", sizes[1]->x, sizes[1]->y, sizes[1]->z);

    int roundsSize = DATA_SIZE/rounds;
    for (int i = 0; (i < 1)/* && (changed)*/; ++i){
        printf("\nEntered #%d kernel outer-loop\n", i+1);
        for (int j = 0; j < rounds; ++j){
            //printf("Iteration #%d of inner-loop\n", j+1);
            gpuGRKCall(sizes, &changed, gpu_changed, cudaImage, cudaRegion, (roundsSize*j));
        }
        if (0 != DATA_SIZE%rounds){
            printf("DATA_SIZE%%rounds != 0, running %dth iteration of inner-loop\n", rounds);
            gpuGRKCall(sizes, &changed, gpu_changed, cudaImage, cudaRegion, (DATA_SIZE-roundsSize-1));
        }
        printf("Finished iteration %d of kernel outer-loop!\n", i+1);
    }

    //Copy region from device
    createCudaEvent(&start);
    gEC(hipMemcpy(region, cudaRegion, dataSize, hipMemcpyDeviceToHost));
    createCudaEvent(&end);
    printf("\nCopying region from device took %f ms\n", getCudaEventTime(start, end));

    gEC(hipFree(cudaImage));
    gEC(hipFree(cudaRegion));
    gEC(hipFree(gpu_changed));

    return region;
}

__global__ void raycast_kernel(unsigned char* data, unsigned char* image, unsigned char* region){
    //blah
    return;
}

unsigned char* raycast_gpu(unsigned char* data, unsigned char* region){

    return NULL;
}

int main(int argc, char** argv){
    //float ms_time;
    /*print_properties();

    printf("Done printing properties\n");*/

    unsigned char* data = create_data();

    printf("Done creating data\n");


    unsigned char* region = grow_region_gpu(data);
    //printf("grow_region_gpu() took %f ms\n", ms_time);

    printf("Done creating region\n");

    unsigned char* image = raycast_gpu(data, region);
    /*printf("raycast_gpu() took %f ms\n", ms_time);*/

    printf("Done creating image\n");

    write_bmp(image, IMAGE_DIM, IMAGE_DIM, "raycast_gpu_out.bmp");

    printf("Done with program\n");
    return 0;
}
