#include "hip/hip_runtime.h"
#include "raycast.h"

__global__ void region_grow_kernel(unsigned char* data, unsigned char* region, int* changed){
    int dx[6] = {-1,1,0,0,0,0};
    int dy[6] = {0,0,-1,1,0,0}; int dz[6] = {0,0,0,0,-1,1};
    int3 pixel = {.z = blockIdx.y*IMAGE_SIZE,
        .y = blockIdx.x*IMAGE_DIM, .x = threadIdx.x};
    int tid = pixel.z*DATA_DIM*DATA_DIM + pixel.y*DATA_DIM + pixel.x;

    if(NEW_VOX == region[tid]){
        printf("Entered first if!\n");
        int3 pos;
        region[tid] = VISITED;
        for (int i = 0; i < 6; ++i){
            pos = pixel;
            pos.x += dx[i];
            pos.y += dy[i];
            pos.z += dz[i];
            if (//Check that pos pixel is inside image/region
                ((pos.x >= 0 && pos.x < DATA_DIM-1) &&
                (pos.y >= 0 && pos.y < DATA_DIM-1) &&
                (pos.z >= 0 && pos.z < DATA_DIM-1)) &&
                //Check that it's not already been "discovered"
                !region[tid] &&
                //Check that the corresponding color values actually match
            (abs(data[tid] -
            data[pos.z*DATA_DIM*DATA_DIM + pos.y*DATA_DIM + pos.x]) < 1)){
                printf("Found neighbour!\n");
                region[tid] = NEW_VOX;
                atomicAdd(changed, 1);
            }
        }
    }

    //*changed = 1;
    return;
}

unsigned char* grow_region_gpu(unsigned char* data){
    printf("Entered grow_region_gpu!\n");
    int finished = 0, *gpu_finished;
    hipEvent_t start, end;
    dim3 blockDim, gridDim;
    stack_t* stack = new_stack();
    int3 seed = {.x = 50, .y = 300, .z = 300};
    unsigned char* cudaImage, *cudaRegion, *region;
    blockDim.x = 512, gridDim.x = 512, gridDim.y = 512;
    region = (unsigned char*) calloc(DATA_SIZE, sizeof(unsigned char));

    push(stack, seed);
    region[seed.z*IMAGE_SIZE + seed.y*DATA_DIM + seed.x] = NEW_VOX;

    printf("Done preparing variables!\n");

    //Malloc image on cuda device
    gEC(hipMalloc(&cudaImage, sizeof(unsigned char)*DATA_SIZE));
    //Malloc region on cuda device
    gEC(hipMalloc(&cudaRegion, sizeof(unsigned char)*DATA_SIZE));
    gEC(hipMalloc(&gpu_finished, sizeof(int)));

    printf("Done mallocing on CUDA device!\n");

    //Copy image and region over to device
    createCudaEvent(&start);
    gEC(hipMemcpy(cudaImage, data, sizeof(unsigned char)*DATA_SIZE, hipMemcpyHostToDevice));
    gEC(hipMemcpy(cudaRegion, region, sizeof(unsigned char)*DATA_SIZE, hipMemcpyHostToDevice));
    createCudaEvent(&end);
    printf("Copying image and region to device took %f ms\n",
        getCudaEventTime(start, end));



    while(!finished){
        printf("Entered while-loop\n");
        gEC(hipMemcpy(gpu_finished, &finished, sizeof(int), hipMemcpyHostToDevice));

        region_grow_kernel<<<gridDim, blockDim>>>(data, region, gpu_finished);

        gEC(hipMemcpy(&finished, gpu_finished, sizeof(int), hipMemcpyDeviceToHost));
    }


    //Copy region from device
    createCudaEvent(&start);
    gEC(hipMemcpy(region, cudaRegion, sizeof(unsigned char)*DATA_SIZE, hipMemcpyDeviceToHost));
    createCudaEvent(&end);
    printf("Copying region from device took %f ms\n", getCudaEventTime(start, end));

    return region;
}

__global__ void raycast_kernel(unsigned char* data, unsigned char* image, unsigned char* region){
    //blah
    return;
}

unsigned char* raycast_gpu(unsigned char* data, unsigned char* region){

    return NULL;
}

int main(int argc, char** argv){
    //float ms_time;
    /*print_properties();

    printf("Done printing properties\n");

    printf("size of data: %zd\n", sizeof(unsigned char)*DATA_DIM*DATA_DIM*DATA_DIM/(1024*1024));*/

    unsigned char* data = create_data();

    printf("Done creating data\n");


    unsigned char* region = grow_region_gpu(data);
    //printf("grow_region_gpu() took %f ms\n", ms_time);

    printf("Done creating region\n");

    unsigned char* image = raycast_gpu(data, region);
    /*printf("raycast_gpu() took %f ms\n", ms_time);*/

    printf("Done creating image\n");

    //write_bmp(image, IMAGE_DIM, IMAGE_DIM, "raycast_gpu_out.bmp");

    printf("Done with program\n");
    return 0;
}
