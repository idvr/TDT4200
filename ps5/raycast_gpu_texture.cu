#include "hip/hip_runtime.h"
#include "raycast.cuh"

// float3 utilities
__host__ __device__ float3 cross(float3 a, float3 b){
    float3 c;
    c.x = a.y*b.z - a.z*b.y;
    c.y = a.z*b.x - a.x*b.z;
    c.z = a.x*b.y - a.y*b.x;
    return c;
}

__host__ __device__ float3 normalize(float3 v){
    float l = sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
    v.x /= l;
    v.y /= l;
    v.z /= l;
    return v;
}

__host__ __device__ float3 add(float3 a, float3 b){
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
    return a;
}

__host__ __device__ float3 scale(float3 a, float b){
    a.x *= b;
    a.y *= b;
    a.z *= b;
    return a;
}

// Trilinear interpolation
__host__ __device__ float value_at(float3 pos, uchar* data){
    if(!inside(pos)){
        return 0;
    }

    int x = floor(pos.x);
    int y = floor(pos.y);
    int z = floor(pos.z);

    int x_u = ceil(pos.x);
    int y_u = ceil(pos.y);
    int z_u = ceil(pos.z);

    float rx = pos.x - x;
    float ry = pos.y - y;
    float rz = pos.z - z;

    float a0 = rx*data[index(z,y,x)] + (1-rx)*data[index(z,y,x_u)];
    float a1 = rx*data[index(z,y_u,x)] + (1-rx)*data[index(z,y_u,x_u)];
    float a2 = rx*data[index(z_u,y,x)] + (1-rx)*data[index(z_u,y,x_u)];
    float a3 = rx*data[index(z_u,y_u,x)] + (1-rx)*data[index(z_u,y_u,x_u)];

    float b0 = ry*a0 + (1-ry)*a1;
    float b1 = ry*a2 + (1-ry)*a3;

    float c0 = rz*b0 + (1-rz)*b1;

    return c0;
}

__device__ int getBlockId_3D(){
    return blockIdx.x + (blockIdx.y*gridDim.x)
            + (blockIdx.z*gridDim.x*gridDim.y);
}

__device__ int getBlockThreadId_3D(){
    return threadIdx.x + (threadIdx.y*blockDim.x)
            + (threadIdx.z*blockDim.x*blockDim.y);
}

__host__ __device__ int index(int3 pos){
    return pos.z*IMAGE_SIZE
        + pos.y*DATA_DIM + pos.x;
}

__host__ __device__ int index(int z, int y, int x){
    return z*IMAGE_SIZE
        + y*DATA_DIM + x;
}

__device__ int getGlobalIdx_3D_3D(){
    int blockId = getBlockId_3D();
    int threadId = getBlockThreadId_3D() +
            blockId*(blockDim.x*blockDim.y*blockDim.z);
    return threadId;
}

__host__ __device__ int3 getGlobalPos(int globalThreadId){
    int3 pos = {
        .x = globalThreadId,
            .y = 0, .z = 0};

    //Check if x > (512^2 - 1)
    if ((IMAGE_SIZE-1) < pos.x){
        pos.z = pos.x/IMAGE_SIZE;
        pos.x -= pos.z*IMAGE_SIZE;
    }

    //Check if x > (512 - 1)
    if ((IMAGE_DIM-1) < pos.x){
        pos.y = pos.x/IMAGE_DIM;
        pos.x -= pos.y*IMAGE_DIM;
    }

    return pos;
}

__host__ __device__ int similar(uchar* data, int3 a, int3 b){
    uchar va = data[index(a)];
    uchar vb = data[index(b)];
    return (abs(va-vb) < 1);
}

__host__ __device__ int inside(int3 pos){
    int x = (pos.x >= 0 && pos.x < DATA_DIM-1);
    int y = (pos.y >= 0 && pos.y < DATA_DIM-1);
    int z = (pos.z >= 0 && pos.z < DATA_DIM-1);
    return x && y && z;
}

__host__ __device__ int inside(float3 pos){
    int x = (pos.x >= 0 && pos.x < DATA_DIM-1);
    int y = (pos.y >= 0 && pos.y < DATA_DIM-1);
    int z = (pos.z >= 0 && pos.z < DATA_DIM-1);
    return x && y && z;
}

// Serial ray casting
uchar* raycast_serial(uchar* data, uchar* region){
    uchar* image = (uchar*)malloc(sizeof(uchar)*IMAGE_SIZE);
    float3 camera = {.x=1000,.y=1000,.z=1000};
    float3 forward = {.x=-1, .y=-1, .z=-1};
    float3 z_axis = {.x=0, .y=0, .z = 1};

    float3 right = cross(forward, z_axis);
    float3 up = cross(right, forward);
    forward = normalize(forward);
    right = normalize(right);
    up = normalize(up);

    float fov = 3.14/4;
    float pixel_width = tan(fov/2.0)/(IMAGE_DIM/2);
    float step_size = 0.5;

    int cntr = 0;
    for(int y = -(IMAGE_DIM/2); y < (IMAGE_DIM/2); y++){
        for(int x = -(IMAGE_DIM/2); x < (IMAGE_DIM/2); x++){
            float3 screen_center = add(camera, forward);
            float3 ray = add(add(screen_center,
                scale(right, x*pixel_width)), scale(up, y*pixel_width));
            ray = add(ray, scale(camera, -1));
            ray = normalize(ray);
            float3 pos = camera;
            int i = 0;
            float color = 0;
            while(color < 255 && i < 5000){
                i++;
                pos = add(pos, scale(ray, step_size));   // Update position
                int r = value_at(pos, region);           // Check if we're in the region
                color += value_at(pos, data)*(0.01 + r) ;// Update the color based on data value, and if we're in the region
            }
            image[(y+(IMAGE_DIM/2))*IMAGE_DIM + (x+(IMAGE_DIM/2))]
                                        = color > 255 ? 255 : color;
        }
        cntr++;
        if (0 == cntr%10){
            printf("Done with image row #%d\n", y+(IMAGE_DIM/2));
        }
    }
    return image;
}

// Serial region growing, same algorithm as in assignment 2
uchar* grow_region_serial(uchar* data){
    uchar* region = (uchar*)calloc(sizeof(uchar), DATA_DIM*DATA_DIM*DATA_DIM);

    stack_t* stack = new_stack();

    int3 seed = {.x=50, .y=300, .z=300};
    push(stack, seed);
    region[seed.z *DATA_DIM*DATA_DIM + seed.y*DATA_DIM + seed.x] = 1;

    int dx[6] = {-1,1,0,0,0,0};
    int dy[6] = {0,0,-1,1,0,0};
    int dz[6] = {0,0,0,0,-1,1};

    while(stack->size > 0){
        int3 pixel = pop(stack);
        for(int n = 0; n < 6; n++){
            int3 candidate = pixel;
            candidate.x += dx[n];
            candidate.y += dy[n];
            candidate.z += dz[n];

            if(!inside(candidate)){
                continue;
            }

            if(region[candidate.z * DATA_DIM*DATA_DIM + candidate.y*DATA_DIM + candidate.x]){
                continue;
            }

            if(similar(data, pixel, candidate)){
                push(stack, candidate);
                region[candidate.z * DATA_DIM*DATA_DIM + candidate.y*DATA_DIM + candidate.x] = 1;
            }
        }
    }

    return region;
}

__global__ void region_grow_kernel(uchar* data, uchar* region, int* changed){
    const int dx[6] = {-1,1,0,0,0,0};
    const int dy[6] = {0,0,-1,1,0,0};
    const int dz[6] = {0,0,0,0,-1,1};
    int tid = getGlobalIdx_3D_3D();
    int3 pixel = getGlobalPos(tid);

    if(NEW_VOX == region[tid]){
        int3 pos; int pos_id;
        region[tid] = VISITED;
        for (int i = 0; i < 6; ++i){
            pos = pixel;
            pos.x += dx[i];
            pos.y += dy[i];
            pos.z += dz[i];
            pos_id = index(pos);
            if (inside(pos)     &&
                !region[pos_id] &&
                abs(data[tid] - data[pos_id]) < 1){
                region[pos_id] = NEW_VOX;
                *changed = 1;
            }
        }
    }
    return;
}

uchar* grow_region_gpu(uchar* data){
    hipEvent_t start, end;
    int changed = 1, *gpu_changed;
    stack2_t *time_stack = new_time_stack(175);
    dim3 **sizes = getGridsBlocksGrowRegion(0);
    int3 seed = {.x = 50, .y = 300, .z = 300};
    uchar *cudaData, *cudaRegion, *region;

    region = (uchar*) calloc(sizeof(uchar), DATA_SIZE);
    region[seed.z*IMAGE_SIZE + seed.y*DATA_DIM + seed.x] = NEW_VOX;
    //printf("Done instantiating variables...\n");

    gEC(hipMalloc(&gpu_changed, sizeof(int)));
    //Malloc image on cuda device
    gEC(hipMalloc(&cudaData, dataSize));
    //Malloc region on cuda device
    gEC(hipMalloc(&cudaRegion, dataSize));
    gEC(hipMemset(cudaRegion, 0, dataSize));
    //printf("Done mallocing on CUDA device!\n");

    //Copy image and region over to device
    createCudaEvent(&start);
    gEC(hipMemcpy(cudaData, data, dataSize, hipMemcpyHostToDevice));
    gEC(hipMemcpy(cudaRegion, region, dataSize, hipMemcpyHostToDevice));
    createCudaEvent(&end);
    printf("Copying data and region to device took %f ms\n\n",
        getCudaEventTime(start, end));

    for (int i = 0; changed && (175 > i); ++i){
        gEC(hipMemset(gpu_changed, 0, sizeof(int)));
        createCudaEvent(&start);
        region_grow_kernel<<<*sizes[0], *sizes[1]>>>(&cudaData[0], &cudaRegion[0], gpu_changed);
        createCudaEvent(&end);
        push(time_stack, getCudaEventTime(start, end));
        gEC(hipMemcpy(&changed, gpu_changed, sizeof(int), hipMemcpyDeviceToHost));
    }

    float sum = 0;
    for (int i = 0; i < time_stack->size; ++i){
        sum += peek(time_stack, i);
    }
    printf("%d kernel calls took a sum total of %f ms\n\n", time_stack->size, sum);
    destroy(time_stack);

    //Copy region from device
    createCudaEvent(&start);
    gEC(hipMemcpy(region, cudaRegion, dataSize, hipMemcpyDeviceToHost));
    createCudaEvent(&end);
    printf("\nCopying region from device took %f ms\n\n", getCudaEventTime(start, end));

    gEC(hipFree(cudaData));
    gEC(hipFree(cudaRegion));
    gEC(hipFree(gpu_changed));

    return region;
}

__global__ void raycast_kernel(uchar* data, uchar* image, uchar* region){
    int tid = getGlobalIdx_3D_3D();
    int y = getBlockId_3D() - (IMAGE_DIM/2);
    int x = getBlockThreadId_3D() - (IMAGE_DIM/2);
    float3 z_axis = {.x=0, .y=0, .z = 1};
    float3 forward = {.x=-1, .y=-1, .z=-1};
    float3 camera = {.x=1000, .y=1000, .z=1000};
    float3 right = cross(forward, z_axis);
    float3 up = cross(right, forward);

    float fov = 3.14/4;
    up = normalize(up);
    float step_size = 0.5;
    right = normalize(right);
    forward = normalize(forward);
    float pixel_width = tan(fov/2.0)/(IMAGE_DIM/2);

    //Do the raycasting
    float3 screen_center = add(camera, forward);
    float3 ray = add(add(screen_center,
        scale(right, x*pixel_width)), scale(up, y*pixel_width));
    ray = add(ray, scale(camera, -1));
    ray = normalize(ray);
    float3 pos = camera;

    float color = 0;
    for (int i = 0; 255 > color && (5000 > i); ++i){
        pos = add(pos, scale(ray, step_size));
        int r = value_at(pos, region);
        color += value_at(pos, data)*(0.01+r);
    }
    image[tid] = min(color, 255.f);
    return;
}

uchar* raycast_gpu(uchar* data, uchar* region){
    hipEvent_t start, end;
    dim3 **sizes = getGridsBlocksRaycasting(0);
    uchar *cudaImage, *cudaRegion, *cudaData;
    uchar *image = (uchar*) malloc(imageSize);

    //Malloc image++ on cuda device
    gEC(hipMalloc(&cudaData, dataSize));
    gEC(hipMalloc(&cudaImage, imageSize));
    gEC(hipMalloc(&cudaRegion, dataSize));
    gEC(hipMemset(cudaImage, 0, imageSize));
    //printf("Done mallocing on CUDA device!\n");

    //Copy data and region over to device
    createCudaEvent(&start);
    gEC(hipMemcpy(cudaData, data, dataSize, hipMemcpyHostToDevice));
    gEC(hipMemcpy(cudaRegion, region, dataSize, hipMemcpyHostToDevice));
    createCudaEvent(&end);
    printf("Copying data and region to device took %f ms\n\n",
        getCudaEventTime(start, end));

    createCudaEvent(&start);
    raycast_kernel<<<*sizes[0], *sizes[1]>>>(cudaData, cudaImage, cudaRegion);
    createCudaEvent(&end);
    printf("Calling kernel took %f ms\n", getCudaEventTime(start, end));

    //Copy image back from device
    createCudaEvent(&start);
    gEC(hipMemcpy(image, cudaImage, imageSize, hipMemcpyDeviceToHost));
    createCudaEvent(&end);
    printf("Copying image from device took %f ms\n\n",
        getCudaEventTime(start, end));

    gEC(hipFree(cudaData));
    gEC(hipFree(cudaImage));
    gEC(hipFree(cudaRegion));
    return image;
}

__device__ float valueAtData(float3 pos){
    if (!inside(pos)){
        return 0;
    }
    return tex3D(data_texture, pos.x, pos.y, pos.z);
}

__device__ float valueAtRegion(float3 pos){
    if (!inside(pos)){
        return 0;
    }
    return tex3D(region_texture, pos.x, pos.y, pos.z);
}

__global__ void raycast_kernel_texture(uchar* image){
    int tid = getGlobalIdx_3D_3D();
    int y = getBlockId_3D() - (IMAGE_DIM/2);
    int x = getBlockThreadId_3D() - (IMAGE_DIM/2);
    float step_size = 0.5, fov = 3.14/4, color = 0,
            pixel_width = tan(fov/2.0)/(IMAGE_DIM/2);
    float3 z_axis = {.x=0, .y=0, .z = 1};
    float3 forward = {.x=-1, .y=-1, .z=-1};
    float3 camera = {.x=1000, .y=1000, .z=1000};

    float3 right = cross(forward, z_axis);
    float3 up = cross(right, forward);

    up = normalize(up);
    right = normalize(right);
    forward = normalize(forward);

    float3 screen_center = add(camera, forward);
    float3 ray = add(add(screen_center,
        scale(right, x*pixel_width)), scale(up, y*pixel_width));
    ray = add(ray, scale(camera, -1));
    ray = normalize(ray);
    float3 pos = camera;

    for (int i = 0; 255 > color && 5000 > i; ++i){
        pos = add(pos, scale(ray, step_size));
        int r = valueAtRegion(pos);
        color += valueAtData(pos)*(0.01+r);
    }
    image[tid] = min(color, 255.f);
}

uchar* raycast_gpu_texture(uchar* data, uchar* region){
    printf("Entered gpu_texture()\n");
    hipEvent_t start, end;
    uchar *cudaImage;
    hipArray *cudaData, *cudaRegion;
    dim3 **sizes = getGridsBlocksRaycasting(0);
    uchar *image = (uchar*) malloc(imageSize);
    hipMemcpy3DParms copyData = {0}, copyRegion = {0};
    const hipExtent volumeSize = make_hipExtent(DATA_DIM, DATA_DIM, DATA_DIM);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    printf("Finished creating variables.\n");

    gEC(hipMalloc(&cudaImage, imageSize));
    gEC(hipMemset(cudaImage, 0, imageSize));
    gEC(hipMalloc3DArray(&cudaData, &channelDesc, volumeSize));
    gEC(hipMalloc3DArray(&cudaRegion, &channelDesc, volumeSize));

    //For data
    copyData.dstArray = cudaData;
    copyData.extent = volumeSize;
    copyData.kind = hipMemcpyHostToDevice;
    copyData.srcPtr = make_hipPitchedPtr((void*)data,
        volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);

    //For region
    copyRegion.extent = volumeSize;
    copyRegion.dstArray = cudaRegion;
    copyRegion.kind = hipMemcpyHostToDevice;
    copyRegion.srcPtr = make_hipPitchedPtr((void*)region,
        volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);

    data_texture.normalized = true;
    data_texture.filterMode = hipFilterModeLinear;
    data_texture.addressMode[0] = hipAddressModeBorder;
    data_texture.addressMode[1] = hipAddressModeBorder;
    data_texture.addressMode[2] = hipAddressModeBorder;

    region_texture.normalized = true;
    region_texture.filterMode = hipFilterModeLinear;
    region_texture.addressMode[0] = hipAddressModeBorder;
    region_texture.addressMode[1] = hipAddressModeBorder;
    region_texture.addressMode[2] = hipAddressModeBorder;
    printf("cuda variables/structs set up.\n");

    createCudaEvent(&start);
    gEC(hipMemcpy3D(&copyData));
    gEC(hipMemcpy3D(&copyRegion));
    gEC(hipBindTextureToArray(data_texture, cudaData, channelDesc));
    gEC(hipBindTextureToArray(region_texture, cudaRegion, channelDesc));
    createCudaEvent(&end);
    printf("Copying and binding data and region to textures took %f ms\n",
        getCudaEventTime(start, end));

    createCudaEvent(&start);
    raycast_kernel_texture<<<*sizes[0], *sizes[1]>>>(cudaImage);
    createCudaEvent(&end);
    printf("Calling kernel took %f ms\n", getCudaEventTime(start, end));

    //Copy image back from device
    createCudaEvent(&start);
    gEC(hipMemcpy(image, cudaImage, imageSize, hipMemcpyDeviceToHost));
    createCudaEvent(&end);
    printf("Copying image from device took %f ms\n\n",
        getCudaEventTime(start, end));

    gEC(hipFreeArray(cudaData));
    gEC(hipFree(cudaImage));
    gEC(hipFreeArray(cudaRegion));
    return image;
}

int main(int argc, char** argv){
    //print_properties();

    uchar* data = create_data();
    printf("Done creating data\n");

    //Serial version
    //uchar* region = grow_region_serial(data);
    uchar* region = grow_region_gpu(data);
    printf("Done creating region\n");

    //Serial version
    //uchar* image = raycast_serial(data, region);
    uchar* image = raycast_gpu_texture(data, region);
    //uchar* image = raycast_gpu(data, region);
    printf("Done creating image\n");

    write_bmp(image, IMAGE_DIM, IMAGE_DIM, "raycast_gpu_texture_out.bmp");
    printf("Done with program\n");

    return 0;
}
