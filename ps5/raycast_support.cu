#include "hip/hip_runtime.h"
#include "raycast.h"

stack_t* new_stack(){
    stack_t* stack = (stack_t*)malloc(sizeof(stack_t));
    stack->size = 0;
    stack->buffer_size = 1024;
    stack->pixels = (int3*)malloc(sizeof(int3)*1024);

    return stack;
}

void push(stack_t* stack, int3 p){
    if(stack->size == stack->buffer_size){
        stack->buffer_size *= 2;
        int3* temp = stack->pixels;
        stack->pixels = (int3*)malloc(sizeof(int3)*stack->buffer_size);
        memcpy(stack->pixels, temp, sizeof(int3)*stack->buffer_size/2);
        free(temp);

    }
    stack->pixels[stack->size] = p;
    stack->size += 1;
}

int3 pop(stack_t* stack){
    stack->size -= 1;
    return stack->pixels[stack->size];
}

// float3 utilities
float3 cross(float3 a, float3 b){
    float3 c;
    c.x = a.y*b.z - a.z*b.y;
    c.y = a.z*b.x - a.x*b.z;
    c.z = a.x*b.y - a.y*b.x;

    return c;
}

float3 normalize(float3 v){
    float l = sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
    v.x /= l;
    v.y /= l;
    v.z /= l;

    return v;
}

float3 add(float3 a, float3 b){
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;

    return a;
}

float3 scale(float3 a, float b){
    a.x *= b;
    a.y *= b;
    a.z *= b;

    return a;
}

// Prints CUDA device properties
void print_properties(){
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    printf("Device count: %d\n\n", deviceCount);

    hipDeviceProp_t p;
    for (int i = 0; i < deviceCount; ++i){
        int cudaReturnStatus = hipSetDevice(i);
        if (hipSuccess != cudaReturnStatus){
            printf("hipSetDevice(%d) returned error\n", i);
            continue;
        }

        cudaReturnStatus = hipGetDeviceProperties (&p, i);
        if (hipSuccess != cudaReturnStatus){
            printf("hipGetDeviceProperties(&p, %d) returned error\n", i);
            continue;
        }

        //If all went well, print info:
        printf("Device #%d, Name: %s\n" , (i+1), p.name);
        printf("Compute capability: %d.%d\n", p.major, p.minor);

        printf("Total memory: %zdGiB \nShared memory per block: %zdKiB\n", p.totalGlobalMem/(1024*1024*1024), p.sharedMemPerBlock/1024);

        printf("#Threads per Warp: %d\n", p.warpSize);

        printf("Multiprocessor (SM/SMX) count: %d\n", p.multiProcessorCount);

        printf("Max threads per Blocks: ");
        for (int j = 0; j < 2; ++j){
            printf("%d, ", p.maxThreadsDim[j]);
        }printf("%d\n", p.maxThreadsDim[2]);

        printf("Max Grid Size: ");
        for (int j = 0; j < 2; ++j){
            printf("%d, ", p.maxGridSize[j]);
        }printf("%d\n", p.maxGridSize[2]);

        printf("Max Threads per Block: %d\n", p.maxThreadsPerBlock);

        printf("Are concurrent kernels supported?: %s\n", p.concurrentKernels ? "yes" : "no");

        printf("\n\n");
    }
}

// Fills data with values
unsigned char func(int x, int y, int z){
    unsigned char value = rand() % 20;

    int x1 = 300;
    int y1 = 400;
    int z1 = 100;
    float dist = sqrt((x-x1)*(x-x1) + (y-y1)*(y-y1) + (z-z1)*(z-z1));

    if(dist < 100){
        value  = 30;
    }

    x1 = 100;
    y1 = 200;
    z1 = 400;
    dist = sqrt((x-x1)*(x-x1) + (y-y1)*(y-y1) + (z-z1)*(z-z1));

    if(dist < 50){
        value = 50;
    }

    if(x > 200 && x < 300 && y > 300 && y < 500 && z > 200 && z < 300){
        value = 45;
    }
    if(x > 0 && x < 100 && y > 250 && y < 400 && z > 250 && z < 400){
        value =35;
    }
    return value;
}

unsigned char* create_data(){
    unsigned char* data = (unsigned char*)malloc(sizeof(unsigned char) * DATA_DIM*DATA_DIM*DATA_DIM);

    for(int i = 0; i < DATA_DIM; i++){
        for(int j = 0; j < DATA_DIM; j++){
            for(int k = 0; k < DATA_DIM; k++){
                data[i*DATA_DIM*DATA_DIM + j*DATA_DIM+ k]= func(k,j,i);
            }
        }
    }

    return data;
}

// Checks if position is inside the volume (float3 and int3 versions)
int inside(float3 pos){
    int x = (pos.x >= 0 && pos.x < DATA_DIM-1);
    int y = (pos.y >= 0 && pos.y < DATA_DIM-1);
    int z = (pos.z >= 0 && pos.z < DATA_DIM-1);

    return x && y && z;
}

int inside(int3 pos){
    int x = (pos.x >= 0 && pos.x < DATA_DIM);
    int y = (pos.y >= 0 && pos.y < DATA_DIM);
    int z = (pos.z >= 0 && pos.z < DATA_DIM);

    return x && y && z;
}

// Indexing function (note the argument order)
int index(int z, int y, int x){
    return z * DATA_DIM*DATA_DIM + y*DATA_DIM + x;
}

// Trilinear interpolation
float value_at(float3 pos, unsigned char* data){
    if(!inside(pos)){
        return 0;
    }

    int x = floor(pos.x);
    int y = floor(pos.y);
    int z = floor(pos.z);

    int x_u = ceil(pos.x);
    int y_u = ceil(pos.y);
    int z_u = ceil(pos.z);

    float rx = pos.x - x;
    float ry = pos.y - y;
    float rz = pos.z - z;

    float a0 = rx*data[index(z,y,x)] + (1-rx)*data[index(z,y,x_u)];
    float a1 = rx*data[index(z,y_u,x)] + (1-rx)*data[index(z,y_u,x_u)];
    float a2 = rx*data[index(z_u,y,x)] + (1-rx)*data[index(z_u,y,x_u)];
    float a3 = rx*data[index(z_u,y_u,x)] + (1-rx)*data[index(z_u,y_u,x_u)];

    float b0 = ry*a0 + (1-ry)*a1;
    float b1 = ry*a2 + (1-ry)*a3;

    float c0 = rz*b0 + (1-rz)*b1;


    return c0;
}

// Check if two values are similar, threshold can be changed.
int similar(unsigned char* data, int3 a, int3 b){
    unsigned char va = data[a.z * DATA_DIM*DATA_DIM + a.y*DATA_DIM + a.x];
    unsigned char vb = data[b.z * DATA_DIM*DATA_DIM + b.y*DATA_DIM + b.x];

    int i = abs(va-vb) < 1;
    return i;
}
