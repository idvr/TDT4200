#include "hip/hip_runtime.h"
#include "raycast.cuh"

__device__
int inside(int3 pos){
    int x = (pos.x >= 0 && pos.x < DATA_DIM-1);
    int y = (pos.y >= 0 && pos.y < DATA_DIM-1);
    int z = (pos.z >= 0 && pos.z < DATA_DIM-1);
    return x && y && z;
}

__device__
int inside(float3 pos){
    int x = (pos.x >= 0 && pos.x < DATA_DIM-1);
    int y = (pos.y >= 0 && pos.y < DATA_DIM-1);
    int z = (pos.z >= 0 && pos.z < DATA_DIM-1);
    return x && y && z;
}

// float3 utilities
__device__
float3 normalize(float3 v){
    float l = sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
    v.x /= l;
    v.y /= l;
    v.z /= l;
    return v;
}

__device__
float3 add(float3 a, float3 b){
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
    return a;
}

__device__
float3 scale(float3 a, float b){
    a.x *= b;
    a.y *= b;
    a.z *= b;
    return a;
}

__device__
float3 cross(float3 a, float3 b){
    float3 c;
    c.x = a.y*b.z - a.z*b.y;
    c.y = a.z*b.x - a.x*b.z;
    c.z = a.x*b.y - a.y*b.x;
    return c;
}

__device__
int3 getThreadPosInBlock(){
    int3 pos = {
        .x = threadIdx.x,
        .y = threadIdx.y,
        .z = threadIdx.z};
    return pos;
}

__device__
int getThreadInBlockIndex(int3 pos){
    if (!insideThreadBlock(pos)){
        return 0;
    }
    return pos.x +
        (pos.y*blockDim.x) +
        (pos.z*blockDim.x*blockDim.y);
}

__device__
int insideThreadBlock(int3 pos){
    int x = (pos.x >= 0 && pos.x < blockDim.x);
    int y = (pos.y >= 0 && pos.y < blockDim.y);
    int z = (pos.z >= 0 && pos.z < blockDim.z);
    return x && y && z;
}

__device__
int index(int3 pos){
    return pos.z*IMAGE_SIZE
        + pos.y*DATA_DIM + pos.x;
}

__device__
int index(int z, int y, int x){
    return z*IMAGE_SIZE
        + y*DATA_DIM + x;
}

__device__
int isOnEdgeOfThreadBlock(int3 voxel){
    //Check if thread is along one border-edge of the cube or another
    if (0 == voxel.x || //if along plane x == 0
        0 == voxel.y || //if along plane y == 0
        0 == voxel.z || //if along plane z == 0
        (blockDim.x-1 == voxel.x)|| //if along plane x == max value
        (blockDim.y-1 == voxel.y)|| //if along plane y == max value
        (blockDim.z-1 == voxel.z)){ //if along plane z == max value
        return 1;
    }
    return 0;
}

__device__
int3 getGlobalPos(int globalThreadId){
    int3 pos = {
        .x = globalThreadId,
            .y = 0, .z = 0};

    //Check if x > (512^2 - 1)
    if ((IMAGE_SIZE-1) < pos.x){
        pos.z = pos.x/IMAGE_SIZE;
        pos.x -= pos.z*IMAGE_SIZE;
    }

    //Check if x > (512 - 1)
    if ((IMAGE_DIM-1) < pos.x){
        pos.y = pos.x/IMAGE_DIM;
        pos.x -= pos.y*IMAGE_DIM;
    }

    return pos;
}

__device__
int getBlockId(){
    return (blockIdx.x +
        (blockIdx.y*gridDim.x) +
        (blockIdx.z*gridDim.x*gridDim.y));
}

__device__
int getThreadId(){
    return threadIdx.x +
        (threadIdx.y*blockDim.x) +
        (threadIdx.z*blockDim.x*blockDim.y);
}

__device__
int getGlobalIdx(){
    return getThreadId() +
        (getBlockId() *
        (blockDim.x*blockDim.y*blockDim.z));
}

__device__
int similar(uchar* data, int idx, int idy){
    uchar va = data[idx];
    uchar vb = data[idy];
    return (abs(va-vb) < 1);
}

__global__
void raycast_kernel_texture(uchar* image){
    int x = threadIdx.x + (blockIdx.x*blockDim.x);
    int y = threadIdx.y + (blockIdx.y*blockDim.y);
    int tid = x + y*IMAGE_DIM;
    x -= IMAGE_DIM/2; y -= IMAGE_DIM/2;
    float step_size = 0.5, fov = 3.14/4, color = 0,
        pixel_width = tan(fov/2.0)/(IMAGE_DIM/2);
    float3 z_axis = {.x=0, .y=0, .z = 1};
    float3 forward = {.x=-1, .y=-1, .z=-1};
    float3 camera = {.x=1000, .y=1000, .z=1000};

    float3 right = cross(forward, z_axis);
    float3 up = cross(right, forward);

    up = normalize(up);
    right = normalize(right);
    forward = normalize(forward);

    float3 screen_center = add(camera, forward);
    float3 ray = add(add(screen_center,
        scale(right, x*pixel_width)), scale(up, y*pixel_width));
    ray = add(ray, scale(camera, -1));
    ray = normalize(ray);
    float3 pos = camera;

    for (int i = 0; 255 > color && 5000 > i; ++i){
        pos = add(pos, scale(ray, step_size));
        if(!inside(pos)){
            continue;
        }
        int r = tex3D(region_texture, pos.x, pos.y, pos.z)*255.f;
        color += tex3D(data_texture, pos.x, pos.y, pos.z)*255.f*(0.01+r);
    }
    image[tid] = min(color, 255.f);
}

uchar* raycast_gpu_texture(uchar* data, uchar* region){
    hipEvent_t start, end;
    uchar *cudaImage;
    hipArray *cudaData, *cudaRegion;
    dim3 **sizes = getGridsBlocksRaycasting(0);
    uchar *image = (uchar*) malloc(imageSize);
    hipMemcpy3DParms copyData = {0}, copyRegion = {0};
    const hipExtent volumeSize = make_hipExtent(DATA_DIM, DATA_DIM, DATA_DIM);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    //printf("Finished creating variables.\n");

    gEC(hipMalloc(&cudaImage, imageSize));
    gEC(hipMemset(cudaImage, 0, imageSize));
    gEC(hipMalloc3DArray(&cudaData, &channelDesc, volumeSize));
    gEC(hipMalloc3DArray(&cudaRegion, &channelDesc, volumeSize));

    //For data
    copyData.dstArray = cudaData;
    copyData.extent = volumeSize;
    copyData.kind = hipMemcpyHostToDevice;
    copyData.srcPtr = make_hipPitchedPtr(data,
        volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);

    //For region
    copyRegion.extent = volumeSize;
    copyRegion.dstArray = cudaRegion;
    copyRegion.kind = hipMemcpyHostToDevice;
    copyRegion.srcPtr = make_hipPitchedPtr(region,
        volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);

    data_texture.normalized = false;
    data_texture.filterMode = hipFilterModeLinear;
    data_texture.addressMode[0] = hipAddressModeBorder;
    data_texture.addressMode[1] = hipAddressModeBorder;
    data_texture.addressMode[2] = hipAddressModeBorder;

    region_texture.normalized = false;
    region_texture.filterMode = hipFilterModeLinear;
    region_texture.addressMode[0] = hipAddressModeBorder;
    region_texture.addressMode[1] = hipAddressModeBorder;
    region_texture.addressMode[2] = hipAddressModeBorder;
    //printf("Texture variables/structs set up.\n");

    createCudaEvent(&start);
    gEC(hipMemcpy3D(&copyData));
    gEC(hipMemcpy3D(&copyRegion));
    gEC(hipBindTextureToArray(data_texture, cudaData, channelDesc));
    gEC(hipBindTextureToArray(region_texture, cudaRegion, channelDesc));
    createCudaEvent(&end);
    printf("Copying and binding data and region to textures took %.4f ms\n",
        getCudaEventTime(start, end));

    createCudaEvent(&start);
    raycast_kernel_texture<<<*sizes[0], *sizes[1]>>>(cudaImage);
    if (hipSuccess != hipGetLastError()){
        printf("Kernel error:\n%s\n\n",
            hipGetErrorString(hipGetLastError()));
    }
    createCudaEvent(&end);
    printf("Calling kernel took %.4f ms\n", getCudaEventTime(start, end));

    //Copy image back from device
    createCudaEvent(&start);
    gEC(hipMemcpy(image, cudaImage, imageSize, hipMemcpyDeviceToHost));
    createCudaEvent(&end);
    printf("Copying image from device took %.4f ms\n",
        getCudaEventTime(start, end));

    gEC(hipFree(cudaImage));
    gEC(hipFreeArray(cudaData));
    gEC(hipFreeArray(cudaRegion));
    return image;
}

__global__
void region_grow_kernel_shared(uchar* data, uchar* region, int* changed){
    //Check first if all of region is empty, then block can return early
    __shared__ bool isEmpty;
    unsigned int globalIdx = getGlobalIdx();
    isEmpty = true;
    __syncthreads();
    if (region[globalIdx]){
        isEmpty = false;
    }
    __syncthreads();
    if (isEmpty){
        return;
    }

    __shared__ uchar sdata[1024];
    unsigned int tid = getThreadId();

    //Load into shared memory
    sdata[tid] = data[globalIdx];
    __syncthreads();

    const int dx[6] = {-1,1,0,0,0,0};
    const int dy[6] = {0,0,-1,1,0,0};
    const int dz[6] = {0,0,0,0,-1,1};
    int3 blockVox = getThreadPosInBlock();
    int3 globalVox = getGlobalPos(globalIdx);

    //If already discovered or not yet (maybe never) reached; skip it
    if (!inside(globalVox) || NEW_VOX != region[globalIdx]){
        return;
    }
    region[globalIdx] = VISITED;

    for (int i = 0; i < 6; ++i){
        int3 curPos = blockVox;
        int3 curPosGlob = globalVox;
        curPos.x += dx[i];
        curPos.y += dy[i];
        curPos.z += dz[i];
        curPosGlob.x += dx[i];
        curPosGlob.y += dy[i];
        curPosGlob.z += dz[i];

        int curIndex = getThreadInBlockIndex(curPos);
        unsigned int globalIndex = index(curPosGlob);

        //If outside or region != 0; skip it
        if (!inside(curPosGlob) || region[globalIndex]){
            continue;
        }

        //if curPos is a voxel on cube outermost edge(s)
        if (isOnEdgeOfThreadBlock(curPos)){
            //if similar is false for edge voxel
            if(!similar(data, globalIdx, globalIndex)){
                continue;
            }
        } else if (!similar(sdata, tid, curIndex)){
            //If similar == 0 for inner-voxel
            continue;
        }

        region[globalIndex] = NEW_VOX;
        *changed = 1;
    }
}

uchar* grow_region_gpu_shared(uchar* data){
    hipEvent_t start, end;
    int changed = 1, *gpu_changed, itrs = 256;
    dim3 **sizes = getGridsBlocksGrowRegion(0);
    stack2_t *time_stack = new_time_stack(itrs);
    uchar *cudaData, *cudaRegion, *region;

    region = (uchar*) calloc(sizeof(uchar), DATA_SIZE);
    region[300*IMAGE_SIZE + 300*DATA_DIM + 50] = NEW_VOX;
    //printf("Done instantiating variables...\n");

    gEC(hipMalloc(&gpu_changed, sizeof(int)));
    //Malloc image on cuda device
    gEC(hipMalloc(&cudaData, dataSize));
    //Malloc region on cuda device
    gEC(hipMalloc(&cudaRegion, dataSize));
    //printf("Done mallocing on CUDA device!\n");

    //Copy image and region over to device
    createCudaEvent(&start);
    gEC(hipMemcpy(cudaData, data, dataSize, hipMemcpyHostToDevice));
    gEC(hipMemcpy(cudaRegion, region, dataSize, hipMemcpyHostToDevice));
    createCudaEvent(&end);
    printf("Copying data and region to device took %.4f ms\n",
        getCudaEventTime(start, end));

    for (int i = 0; changed && (itrs > i); ++i){
        gEC(hipMemset(gpu_changed, 0, sizeof(int)));
        createCudaEvent(&start);
        region_grow_kernel_shared<<<*sizes[0], *sizes[1]>>>(
            cudaData, cudaRegion, gpu_changed);
        if (hipSuccess != hipGetLastError()){
            printf("Kernel error:\n%s\n\n",
                hipGetErrorString(hipGetLastError()));
        }
        createCudaEvent(&end);
        push(time_stack, getCudaEventTime(start, end));
        gEC(hipMemcpy(&changed, gpu_changed, sizeof(int), hipMemcpyDeviceToHost));
    }

    float sum = 0;
    for (int i = 0; i < time_stack->size; ++i){
        sum += peek(time_stack, i);
    }
    printf("%d kernel calls took a sum total of %.4f ms\n", time_stack->size, sum);
    destroy(time_stack);

    //Copy region from device
    createCudaEvent(&start);
    gEC(hipMemcpy(region, cudaRegion, dataSize, hipMemcpyDeviceToHost));
    createCudaEvent(&end);
    printf("\nCopying region from device took %.4f ms\n", getCudaEventTime(start, end));

    gEC(hipFree(cudaData));
    gEC(hipFree(cudaRegion));
    gEC(hipFree(gpu_changed));

    return region;
}

int main(int argc, char** argv){
    printf("\nStarting program...\n\n");

    uchar* data = create_data();
    printf("Done creating data\n\n");

    uchar* region = grow_region_gpu_shared(data);
    printf("Done creating region\n\n");

    uchar* image = raycast_gpu_texture(data, region);
    printf("Done creating image\n\n");

    write_bmp(image, IMAGE_DIM, IMAGE_DIM, "raycast_gpu_combined_out.bmp");
    printf("Done with program\n\n");

    return 0;
}
